﻿#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include <iostream>
#include <string>
#include <stdio.h>
#include <math.h>

using namespace cv;
using namespace std;

int main()
{

	Mat img, dst;
	float tmp1 = 0, tmp2 = 0, d1, d2;
	int cnt = 0;

	img = imread("./input/sword.png");
	dst = Mat::zeros(512, 512, img.type());
	//768

	for (int i = 0; i < img.rows; i++) {
		for (int j = 0; j < img.cols; j++) {
			for (int k = 0; k < 3; k++) {
				dst.at<Vec3b>(i * 1.5, j * 1.5)[k] = img.at<Vec3b>(i, j)[k];

			}
		}
	}

	for (int i = 1; i < dst.rows - 1; i++) {
		for (int j = 1; j < dst.cols - 1; j++) {
			for (int k = 0; k < 3; k++) {
				if (dst.at<Vec3b>(i, j)[k] == 0) { // (i-move vertically, j-move horizontally)

					for (int s = 1; s < dst.rows - 1; s++) { // First we look for pixels left and right, and take those two values if there are
						if (i - s >= 0 && i + s <= dst.rows - 1) {
							if (dst.at<Vec3b>(i - s, j)[k] != 0) { // Check left pixel
								if (tmp1 == 0) {
									tmp1 = dst.at<Vec3b>(i - s, j)[k];
									d1 = s;
								}
							}
							if (dst.at<Vec3b>(i + s, j)[k] != 0) { // Check right pixel
								if (tmp2 == 0) {
									tmp2 = dst.at<Vec3b>(i + s, j)[k];
									d2 = s;
								}
							}
						}
						if (tmp1 != 0 && tmp2 != 0) { // Weighted by the percentage of distance away when all the left and right pixel values are found
							dst.at<Vec3b>(i, j)[k] = tmp1 * (d2 / (d1 + d2)) + tmp2 * (d1 / (d1 + d2));
							tmp1 = 0;
							tmp2 = 0;
							printf("%d / CHECK Running...\n", cnt++);
							break; // No need to find anymore, go to next line
						}
					}
				}

			}
		}
	}

	tmp1 = 0;
	tmp2 = 0;

	for (int i = 1; i < dst.rows - 1; i++) { // This time using the value found above on the line with no pixels left and right
											 // Find neighboring pixels up and down and fill the empty pixels as above
		for (int j = 1; j < dst.cols - 1; j++) {
			for (int k = 0; k < 3; k++) {
				if (dst.at<Vec3b>(i, j)[k] == 0) {

					for (int s = 1; s < dst.rows - 1; s++) {
						if (j - s >= 0 && j + s <= dst.rows - 1) {
							if (dst.at<Vec3b>(i, j - s)[k] != 0) { // Upward direction search
								if (tmp1 == 0) {
									tmp1 = dst.at<Vec3b>(i, j - s)[k];
									d1 = s;
								}
							}
							if (dst.at<Vec3b>(i, j + s)[k] != 0) { // Downward Search
								if (tmp2 == 0) {
									tmp2 = dst.at<Vec3b>(i, j + s)[k];
									d2 = s;
								}
							}
						}
						if (tmp1 != 0 && tmp2 != 0) { // When both neighboring pixels are found, weighted and filled with blank pixels
							printf("%d / 976904 RRUNNING...\n", cnt++);
							dst.at<Vec3b>(i, j)[k] = tmp1 * (d2 / (d1 + d2)) + tmp2 * (d1 / (d1 + d2));
							tmp1 = 0;
							tmp2 = 0;
							break;
						}
					}
				}

			}
		}
	}

	imshow("Original", img);
	imshow("Result", dst);

	waitKey(0);
	return 0;

}